
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
	int i,x,y;
	fprintf(fp,"P3\n");
	fprintf(fp,"%d %d\n",xdim, ydim);
	fprintf(fp,"255\n");
	for (y=0;y<ydim;y++) {
		for (x=0;x<xdim;x++) {
			i=x+y*xdim;
			fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
		}
		fprintf(fp,"\n");
	}
}

int main (void) {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1, 1>>> (d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("c : %d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}