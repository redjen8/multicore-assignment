#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};


__global__ void kernel_compute(Sphere* s, unsigned char* ptr)
{
	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = tx + ty*DIM;
	float ox = (tx - DIM/2);
	float oy = (ty - DIM/2);
	// printf("x:%d, y:%d, ox:%f, oy:%f\n",tx,ty,ox,oy);
	float r=0, g=0, b=0;
	float maxz = -INF;
	for(int i=0; i<SPHERES; i++) {
		float n;
		float   t = s[i].hit( ox, oy, &n );

		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		} 
	}

	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
	int i,x,y;
	fprintf(fp,"P3\n");
	fprintf(fp,"%d %d\n",xdim, ydim);
	fprintf(fp,"255\n");
	for (y=0;y<ydim;y++) {
		for (x=0;x<xdim;x++) {
			i=x+y*xdim;
			fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
		}
		fprintf(fp,"\n");
	}
}

int main (void) {
	unsigned char* bitmap_h;

	srand(time(NULL));

	FILE* fp = fopen("result_cuda.ppm","w");

    clock_t begin = clock();
    // allocate memory for host
	Sphere *sphere_h = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
	for (int i=0; i<SPHERES; i++) {
		sphere_h[i].r = rnd( 1.0f );
		sphere_h[i].g = rnd( 1.0f );
		sphere_h[i].b = rnd( 1.0f );
		sphere_h[i].x = rnd( 2000.0f ) - 1000;
		sphere_h[i].y = rnd( 2000.0f ) - 1000;
		sphere_h[i].z = rnd( 2000.0f ) - 1000;
		sphere_h[i].radius = rnd( 200.0f ) + 40;
	}
	bitmap_h=(unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM*4);

    //allocate memory for device
    int sphereSize = sizeof(Sphere) * SPHERES;
    int bitmapSize = sizeof(unsigned char) * DIM * DIM * 4;
    Sphere* sphere_d = (Sphere*) malloc (sphereSize);
    unsigned char* bitmap_d = (unsigned char*) malloc (bitmapSize);
    hipMalloc((void**)&sphere_d, sphereSize);
    hipMalloc((void**)&bitmap_d, bitmapSize);

    hipMemcpy(sphere_d, sphere_h, sphereSize, hipMemcpyHostToDevice);
    hipMemcpy(bitmap_d, bitmap_h, bitmapSize, hipMemcpyHostToDevice);
	dim3 dimBlock(32, 32);
	dim3 dimGrid(64, 64);
    kernel_compute<<<dimGrid, dimBlock>>>(sphere_d, bitmap_d);
	hipDeviceSynchronize();

	hipMemcpy(bitmap_h, bitmap_d, bitmapSize, hipMemcpyDeviceToHost);

	hipFree(sphere_d);
	hipFree(bitmap_d);

	ppm_write(bitmap_h,DIM,DIM,fp);
    clock_t end = clock();
    double timeDiff = (double) (end-begin) / CLOCKS_PER_SEC;
	printf("Execution time : %lfms\n", timeDiff * 1000);
	printf("[%s] was generated.\n", "result_cuda.ppm");
	fclose(fp);
	free(bitmap_h);
	free(sphere_h);
	return 0;
}