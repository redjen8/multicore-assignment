#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 1000000000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernel_compute() {
    printf("Hello, world!\n");
}

int main(void) {
    kernel_compute<<<10, 1>>>();
    return 0;
}