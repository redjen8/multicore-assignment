#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#define N 1000000000

struct convert_pi
{
	const double step = 0.000000001;
	__device__ 
	double operator() (const int& x) const {
		return 4.0 / (1.0 + ((x + 0.5) * step) * ((x + 0.5) * step));
	}
};

int main(void) {
	double alpha = 1 / (double) N;

	auto startTime = std::chrono::high_resolution_clock::now();
	thrust::device_vector<int> data(N);
	thrust::sequence(data.begin(), data.end());
	convert_pi unary_op;
	thrust::plus<double> binary_op;
	double result = thrust::transform_reduce(thrust::device, data.begin(), data.end(), unary_op, (double) 0.0, binary_op) * alpha;
	hipDeviceSynchronize();
	auto endTime = std::chrono::high_resolution_clock::now();
	auto timeDiff = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
	std::cout.precision(24);
	std::cout << "PI : " << result << std::endl;
	std::cout << "Execution time : " << timeDiff.count() / 1000 << " ms" << std::endl;
	return 0;
}