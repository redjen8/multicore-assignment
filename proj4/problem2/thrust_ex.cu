#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

#define N 1000000000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

double step;

__global__ void kernel_compute() {
    printf("Hello, world!\n");
}

__shared__ int sum;

int main(void) {
    step = 1.0 / (double)N;
    dim3 dimBlock(32, 32);
    dim3 dimGrid(1024, 1024);
    kernel_compute<<<dimGrid, dimBlock>>>();
    gpuErrchk(hipDeviceSynchronize());
    return 0;
}